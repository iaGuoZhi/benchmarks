#include "hip/hip_runtime.h"
/*
GFC code: A GPU-based compressor for arrays of double-precision
floating-point values.

Copyright (c) 2011-2020, Texas State University. All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the following disclaimer in the
     documentation and/or other materials provided with the distribution.
   * Neither the name of Texas State University nor the names of its
     contributors may be used to endorse or promote products derived from
     this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL TEXAS STATE UNIVERSITY BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Authors: Molly A. O'Neil and Martin Burtscher

URL: The latest version of this code is available at
https://userweb.cs.txstate.edu/~burtscher/research/GFC/.

Publication: This work is described in detail in the following paper.
Molly A. O'Neil and Martin Burtscher. Floating-Point Data Compression at 75
Gb/s on a GPU. Proceedings of the Fourth Workshop on General Purpose Processing
Using GPUs, pp. 7:1-7:7. March 2011.
*/


#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define ull unsigned long long
#define MAX (64*1024*1024)

#define WARPSIZE 32

__constant__ int dimensionalityd; // dimensionality parameter
__constant__ ull *cbufd; // ptr to uncompressed data
__constant__ unsigned char *dbufd; // ptr to compressed data
__constant__ ull *fbufd; // ptr to decompressed data
__constant__ int *cutd; // ptr to chunk boundaries
__constant__ int *offd; // ptr to chunk offsets after compression

/************************************************************************************/

/*
This is the GPU compression kernel, which should be launched using the block count
and warps/block:
  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();

Inputs
------
dimensionalityd: dimensionality of trace (from cmd line)
cbufd: ptr to the uncompressed data
cutd: ptr to array of chunk boundaries

Output
------
The compressed data, in dbufd 
Compressed chunk offsets for offset table, in offd
*/

__global__ void CompressionKernel()
{
  register int offset, code, bcount, tmp, off, beg, end, lane, warp, iindex, lastidx, start, term;
  register ull diff, prev;
  __shared__ int ibufs[32 * (3 * WARPSIZE / 2)]; // shared space for prefix sum

  // index within this warp
  lane = threadIdx.x & 31;
  // index within shared prefix sum array
  iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
  ibufs[iindex] = 0;
  iindex += WARPSIZE / 2;
  lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
  // warp id
  warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
  // prediction index within previous subchunk
  offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

  // determine start and end of chunk to compress
  start = 0;
  if (warp > 0) start = cutd[warp-1];
  term = cutd[warp];
  off = ((start+1)/2*17);

  prev = 0;
  for (int i = start + lane; i < term; i += WARPSIZE) {
    // calculate delta between value to compress and prediction
    // and negate if negative
    diff = cbufd[i] - prev;
    code = (diff >> 60) & 8;
    if (code != 0) {
      diff = -diff;
    }

    // count leading zeros in positive delta
    bcount = 8 - (__clzll(diff) >> 3);
    if (bcount == 2) bcount = 3; // encode 6 lead-zero bytes as 5

    // prefix sum to determine start positions of non-zero delta bytes
    ibufs[iindex] = bcount;
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-1];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-2];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-4];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-8];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-16];
    __threadfence_block();

    // write out non-zero bytes of delta to compressed buffer
    beg = off + (WARPSIZE/2) + ibufs[iindex-1];
    end = beg + bcount;
    for (; beg < end; beg++) {
      dbufd[beg] = diff;
      diff >>= 8;
    }

    if (bcount >= 3) bcount--; // adjust byte count for the dropped encoding
    tmp = ibufs[lastidx];
    code |= bcount;
    ibufs[iindex] = code;
    __threadfence_block();

    // write out half-bytes of sign and leading-zero-byte count (every other thread
    // writes its half-byte and neighbor's half-byte)
    if ((lane & 1) != 0) {
      dbufd[off + (lane >> 1)] = ibufs[iindex-1] | (code << 4);
    }
    off += tmp + (WARPSIZE/2);

    // save prediction value from this subchunk (based on provided dimensionality)
    // for use in next subchunk
    prev = cbufd[i + offset];
  }

  // save final value of off, which is total bytes of compressed output for this chunk
  if (lane == 31) offd[warp] = off;
}

/************************************************************************************/

/*
This is the GPU decompression kernel, which should be launched using the block count
and warps/block:
  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();

Inputs
------
dimensionalityd: dimensionality of trace
dbufd: ptr to array of compressed data
cutd: ptr to array of chunk boundaries

Output
------
The decompressed data in fbufd
*/

__global__ void DecompressionKernel()
{
  register int offset, code, bcount, off, beg, end, lane, warp, iindex, lastidx, start, term;
  register ull diff, prev;
  __shared__ int ibufs[32 * (3 * WARPSIZE / 2)];

  // index within this warp
  lane = threadIdx.x & 31;
  // index within shared prefix sum array
  iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
  ibufs[iindex] = 0;
  iindex += WARPSIZE / 2;
  lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
  // warp id
  warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
  // prediction index within previous subchunk
  offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

  // determine start and end of chunk to decompress
  start = 0;
  if (warp > 0) start = cutd[warp-1];
  term = cutd[warp];
  off = ((start+1)/2*17);

  prev = 0;
  for (int i = start + lane; i < term; i += WARPSIZE) {
    // read in half-bytes of size and leading-zero count information
    if ((lane & 1) == 0) {
      code = dbufd[off + (lane >> 1)];
      ibufs[iindex] = code;
      ibufs[iindex + 1] = code >> 4;
    }
    off += (WARPSIZE/2);
    __threadfence_block();
    code = ibufs[iindex];

    bcount = code & 7;
    if (bcount >= 2) bcount++;

    // calculate start positions of compressed data
    ibufs[iindex] = bcount;
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-1];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-2];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-4];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-8];
    __threadfence_block();
    ibufs[iindex] += ibufs[iindex-16];
    __threadfence_block();

    // read in compressed data (the non-zero bytes)
    beg = off + ibufs[iindex-1];
    off += ibufs[lastidx];
    end = beg + bcount - 1;
    diff = 0;
    for (; beg <= end; end--) {
      diff <<= 8;
      diff |= dbufd[end];
    }

    // negate delta if sign bit indicates it was negated during compression
    if ((code & 8) != 0) {
      diff = -diff;
    }

    // write out the uncompressed word
    fbufd[i] = prev + diff;
    __threadfence_block();

    // save prediction for next subchunk
    prev = fbufd[i + offset];
  }
}

/************************************************************************************/

static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

/************************************************************************************/

static void Compress(int blocks, int warpsperblock, int dimensionality)
{
  hipGetLastError();  // reset error value

  // allocate CPU buffers
  ull *cbuf = (ull *)malloc(sizeof(ull) * MAX); // uncompressed data
  if (cbuf == NULL) {
    fprintf(stderr, "cannot allocate cbuf\n"); exit(-1);
  }
  char *dbuf = (char *)malloc(sizeof(char) * ((MAX+1)/2*17)); // compressed data
  if (dbuf == NULL) {
    fprintf(stderr, "cannot allocate dbuf\n"); exit(-1);
  }
  int *cut = (int *)malloc(sizeof(int) * blocks * warpsperblock); // chunk boundaries
  if (cut == NULL) {
    fprintf(stderr, "cannot allocate cut\n"); exit(-1);
  }
  int *off = (int *)malloc(sizeof(int) * blocks * warpsperblock); // offset table
  if (off == NULL) {
    fprintf(stderr, "cannot allocate off\n"); exit(-1);
  }

  // read in trace to cbuf
  int doubles = fread(cbuf, 8, MAX, stdin);

  // calculate required padding for last chunk
  int padding = ((doubles + WARPSIZE - 1) & -WARPSIZE) - doubles;
  doubles += padding;

  // determine chunk assignments per warp
  int per = (doubles + blocks * warpsperblock - 1) / (blocks * warpsperblock);
  if (per < WARPSIZE) per = WARPSIZE;
  per = (per + WARPSIZE - 1) & -WARPSIZE;
  int curr = 0, before = 0, d = 0;
  for (int i = 0; i < blocks * warpsperblock; i++) {
    curr += per;
    cut[i] = min(curr, doubles);
    if (cut[i] - before > 0) {
      d = cut[i] - before;
    }
    before = cut[i];
  }

  // set the pad values to ensure correct prediction
  if (d <= WARPSIZE) {
    for (int i = doubles - padding; i < doubles; i++) {
      cbuf[i] = 0;
    }
  } else {
    for (int i = doubles - padding; i < doubles; i++) {
      cbuf[i] = cbuf[(i & -WARPSIZE) - (dimensionality - i % dimensionality)];
    }
  }

  // allocate GPU buffers
  ull *cbufl; // uncompressed data
  char *dbufl; // compressed data
  int *cutl; // chunk boundaries
  int *offl; // offset table
  if (hipSuccess != hipMalloc((void **)&cbufl, sizeof(ull) * doubles))
    fprintf(stderr, "could not allocate cbufd\n");
  CudaTest("couldn't allocate cbufd");
  if (hipSuccess != hipMalloc((void **)&dbufl, sizeof(char) * ((doubles+1)/2*17)))
    fprintf(stderr, "could not allocate dbufd\n");
  CudaTest("couldn't allocate dbufd");
  if (hipSuccess != hipMalloc((void **)&cutl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate cutd\n");
  CudaTest("couldn't allocate cutd");
  if (hipSuccess != hipMalloc((void **)&offl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate offd\n");
  CudaTest("couldn't allocate offd");

  // copy buffer starting addresses (pointers) and values to constant memory
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dimensionalityd), &dimensionality, sizeof(int)))
    fprintf(stderr, "copying of dimensionality to device failed\n");
  CudaTest("dimensionality copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cbufd), &cbufl, sizeof(void *)))
    fprintf(stderr, "copying of cbufl to device failed\n");
  CudaTest("cbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dbufd), &dbufl, sizeof(void *)))
    fprintf(stderr, "copying of dbufl to device failed\n");
  CudaTest("dbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cutd), &cutl, sizeof(void *)))
    fprintf(stderr, "copying of cutl to device failed\n");
  CudaTest("cutl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(offd), &offl, sizeof(void *)))
    fprintf(stderr, "copying of offl to device failed\n");
  CudaTest("offl copy to device failed");

  // copy CPU buffer contents to GPU
  if (hipSuccess != hipMemcpy(cbufl, cbuf, sizeof(ull) * doubles, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cbuf to device failed\n");
  CudaTest("cbuf copy to device failed");
  if (hipSuccess != hipMemcpy(cutl, cut, sizeof(int) * blocks * warpsperblock, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cut to device failed\n");
  CudaTest("cut copy to device failed");

  CompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();
  CudaTest("compression kernel launch failed");

  // transfer offsets back to CPU
  if(hipSuccess != hipMemcpy(off, offl, sizeof(int) * blocks * warpsperblock, hipMemcpyDeviceToHost))
    fprintf(stderr, "copying of off from device failed\n");
  CudaTest("off copy from device failed");

  // output header
  int num;
  int doublecnt = doubles-padding;
  num = fwrite(&blocks, 1, 1, stdout);
  assert(1 == num);
  num = fwrite(&warpsperblock, 1, 1, stdout);
  assert(1 == num);
  num = fwrite(&dimensionality, 1, 1, stdout);
  assert(1 == num);
  num = fwrite(&doublecnt, 4, 1, stdout);
  assert(1 == num);
  // output offset table
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int start = 0;
    if(i > 0) start = cut[i-1];
    off[i] -= ((start+1)/2*17);
    num = fwrite(&off[i], 4, 1, stdout); // chunk's compressed size in bytes
    assert(1 == num);
  }
  // output compressed data by chunk
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int offset, start = 0;
    if(i > 0) start = cut[i-1];
    offset = ((start+1)/2*17);
    // transfer compressed data back to CPU by chunk
    if (hipSuccess != hipMemcpy(dbuf + offset, dbufl + offset, sizeof(char) * off[i], hipMemcpyDeviceToHost))
      fprintf(stderr, "copying of dbuf from device failed\n");
    CudaTest("dbuf copy from device failed");
    num = fwrite(&dbuf[offset], 1, off[i], stdout);
    assert(off[i] == num);
  }

  free(cbuf);
  free(dbuf);
  free(cut);
  free(off);

  if (hipSuccess != hipFree(cbufl))
    fprintf(stderr, "could not deallocate cbufd\n");
  CudaTest("couldn't deallocate cbufd");
  if (hipSuccess != hipFree(dbufl))
    fprintf(stderr, "could not deallocate dbufd\n");
  CudaTest("couldn't deallocate dbufd");
  if (hipSuccess != hipFree(cutl))
    fprintf(stderr, "could not deallocate cutd\n");
  CudaTest("couldn't deallocate cutd");
  if (hipSuccess != hipFree(offl))
    fprintf(stderr, "could not deallocate offd\n");
  CudaTest("couldn't deallocate offd");
}

/************************************************************************************/

static void Decompress(int blocks, int warpsperblock, int dimensionality, int doubles)
{
  hipGetLastError();  // reset error value

  // allocate CPU buffers
  char *dbuf = (char *)malloc(sizeof(char) * ((MAX+1)/2*17)); // compressed data, divided by chunk
  if (dbuf == NULL) { 
    fprintf(stderr, "cannot allocate dbuf\n"); exit(-1); 
  }
  ull *fbuf = (ull *)malloc(sizeof(ull) * MAX); // decompressed data
  if (fbuf == NULL) { 
    fprintf(stderr, "cannot allocate fbuf\n"); exit(-1);
  }
  int *cut = (int *)malloc(sizeof(int) * blocks * warpsperblock); // chunk boundaries
  if (cut == NULL) { 
    fprintf(stderr, "cannot allocate cut\n"); exit(-1);
  }
  int *off = (int *)malloc(sizeof(int) * blocks * warpsperblock); // offset table
  if(off == NULL) {
    fprintf(stderr, "cannot allocate off\n"); exit(-1);
  }

  // read in offset table
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int num = fread(&off[i], 4, 1, stdin);
    assert(1 == num);
  }

  // calculate required padding for last chunk
  int padding = ((doubles + WARPSIZE - 1) & -WARPSIZE) - doubles;
  doubles += padding;

  // determine chunk assignments per warp
  int per = (doubles + blocks * warpsperblock - 1) / (blocks * warpsperblock); 
  if (per < WARPSIZE) per = WARPSIZE;
  per = (per + WARPSIZE - 1) & -WARPSIZE;
  int curr = 0;
  for (int i = 0; i < blocks * warpsperblock; i++) {
    curr += per;
    cut[i] = min(curr, doubles);
  }

  // allocate GPU buffers
  char *dbufl; // compressed data
  ull *fbufl; // uncompressed data
  int *cutl; // chunk boundaries
  if (hipSuccess != hipMalloc((void **)&dbufl, sizeof(char) * ((doubles+1)/2*17)))
    fprintf(stderr, "could not allocate dbufd\n");
  CudaTest("couldn't allocate dbufd");
  if (hipSuccess != hipMalloc((void **)&fbufl, sizeof(ull) * doubles))
    fprintf(stderr, "could not allocate fbufd\n");
  CudaTest("couldn't allocate fbufd");
  if (hipSuccess != hipMalloc((void **)&cutl, sizeof(int) * blocks * warpsperblock))
    fprintf(stderr, "could not allocate cutd\n");
  CudaTest("couldn't allocate cutd");

  // copy buffer starting addresses (pointers) and values to constant memory
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dimensionalityd), &dimensionality, sizeof(int))) 
    fprintf(stderr, "copying of dimensionality to device failed\n");
  CudaTest("dimensionality copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(dbufd), &dbufl, sizeof(void *)))
    fprintf(stderr, "copying of dbufl to device failed\n");
  CudaTest("dbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(fbufd), &fbufl, sizeof(void *)))
    fprintf(stderr, "copying of fbufl to device failed\n");
  CudaTest("fbufl copy to device failed");
  if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(cutd), &cutl, sizeof(void *)))
    fprintf(stderr, "copying of cutl to device failed\n");
  CudaTest("cutl copy to device failed");

  // read in input data and divide into chunks
  for(int i = 0; i < blocks * warpsperblock; i++) {
    int num, chbeg, start = 0;
    if (i > 0) start = cut[i-1];
    chbeg = ((start+1)/2*17);
    // read in this chunk of data (based on offsets)
    num = fread(&dbuf[chbeg], 1, off[i], stdin);
    assert(off[i] == num);
    // transfer the chunk to the GPU
    if (hipSuccess != hipMemcpy(dbufl + chbeg, dbuf + chbeg, sizeof(char) * off[i], hipMemcpyHostToDevice)) 
      fprintf(stderr, "copying of dbuf to device failed\n");
    CudaTest("dbuf copy to device failed");
  }

  // copy CPU cut buffer contents to GPU
  if (hipSuccess != hipMemcpy(cutl, cut, sizeof(int) * blocks * warpsperblock, hipMemcpyHostToDevice))
    fprintf(stderr, "copying of cut to device failed\n");
  CudaTest("cut copy to device failed");

  DecompressionKernel<<<blocks, WARPSIZE*warpsperblock>>>();
  CudaTest("decompression kernel launch failed");

  // transfer result back to CPU
  if (hipSuccess != hipMemcpy(fbuf, fbufl, sizeof(ull) * doubles, hipMemcpyDeviceToHost))
    fprintf(stderr, "copying of fbuf from device failed\n");
  CudaTest("fbuf copy from device failed");

  // output decompressed data
  int num = fwrite(fbuf, 8, doubles-padding, stdout);
  assert(num == doubles-padding);

  free(dbuf);
  free(fbuf);
  free(cut);

  if(hipSuccess != hipFree(dbufl))
    fprintf(stderr, "could not deallocate dbufd\n");
  CudaTest("couldn't deallocate dbufd");
  if(hipSuccess != hipFree(cutl))
    fprintf(stderr, "could not deallocate cutd\n");
  CudaTest("couldn't deallocate cutd");
}

/************************************************************************************/

static int VerifySystemParameters()
{
  assert(1 == sizeof(char));
  assert(4 == sizeof(int));
  assert(8 == sizeof(ull));
  int val = 1;
  assert(1 == *((char *)&val));

  int current_device = 0, sm_per_multiproc = 0; 
  int max_compute_perf = 0, max_perf_device = 0; 
  int device_count = 0, best_SM_arch = 0; 
  int arch_cores_sm[3] = { 1, 8, 32 }; 
  hipDeviceProp_t deviceProp; 

  hipGetDeviceCount(&device_count);
  if (device_count == 0) {
    fprintf(stderr, "There is no device supporting CUDA\n");
    exit(-1);
  }
   
  // Find the best major SM Architecture GPU device 
  for (current_device = 0; current_device < device_count; current_device++) { 
    hipGetDeviceProperties(&deviceProp, current_device);
    if (deviceProp.major > 0 && deviceProp.major < 9999) { 
      best_SM_arch = max(best_SM_arch, deviceProp.major); 
    }
  }
   
  // Find the best CUDA capable GPU device 
  for (current_device = 0; current_device < device_count; current_device++) { 
    hipGetDeviceProperties(&deviceProp, current_device); 
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
      sm_per_multiproc = 1;
    } 
    else if (deviceProp.major <= 2) { 
      sm_per_multiproc = arch_cores_sm[deviceProp.major]; 
    } 
    else { // Device has SM major > 2 
      sm_per_multiproc = arch_cores_sm[2]; 
    }
      
    int compute_perf = deviceProp.multiProcessorCount * 
                       sm_per_multiproc * deviceProp.clockRate; 
      
    if (compute_perf > max_compute_perf) { 
      // If we find GPU of SM major > 2, search only these 
      if (best_SM_arch > 2) { 
        // If device==best_SM_arch, choose this, or else pass 
        if (deviceProp.major == best_SM_arch) { 
          max_compute_perf = compute_perf; 
          max_perf_device = current_device; 
        } 
      } 
      else { 
        max_compute_perf = compute_perf; 
        max_perf_device = current_device; 
      } 
    } 
  } 
   
  hipGetDeviceProperties(&deviceProp, max_perf_device); 
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "There is no CUDA capable  device\n");
    exit(-1);
  }
  if (deviceProp.major < 2) {
    fprintf(stderr, "Need at least compute capability 2.0\n");
    exit(-1);
  }
  if (deviceProp.warpSize != WARPSIZE) {
    fprintf(stderr, "Warp size must be %d\n", deviceProp.warpSize);
    exit(-1);
  }
  if ((WARPSIZE <= 0) || (WARPSIZE & (WARPSIZE-1) != 0)) {
    fprintf(stderr, "Warp size must be greater than zero and a power of two\n");
    exit(-1);
  }

  return max_perf_device;
}

/************************************************************************************/

int main(int argc, char *argv[])
{
  int blocks, warpsperblock, dimensionality;
  int device;

  fprintf(stderr, "GPU FP Compressor v2.2\n");
  fprintf(stderr, "Copyright 2011-2020 Texas State University\n");

  device = VerifySystemParameters();
  hipSetDevice(device);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(CompressionKernel), hipFuncCachePreferL1);
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(DecompressionKernel), hipFuncCachePreferL1);

  if((3 == argc) || (4 == argc)) { /* compress */
    char dummy;
    blocks = atoi(argv[1]);
    assert((0 < blocks) && (blocks < 256));
    warpsperblock = atoi(argv[2]);
    assert((0 < warpsperblock) && (warpsperblock < 256));
    if(3 == argc) {
      dimensionality = 1;
    } else {
      dimensionality = atoi(argv[3]);
    }
    assert((0 < dimensionality) && (dimensionality <= WARPSIZE));

    Compress(blocks, warpsperblock, dimensionality);
    assert(0 == fread(&dummy, 1, 1, stdin));
  }
  else if(1 == argc) { /* decompress */
    int num, doubles;
    num = fread(&blocks, 1, 1, stdin);
    assert(1 == num);
    blocks &= 255;
    num = fread(&warpsperblock, 1, 1, stdin);
    assert(1 == num);
    warpsperblock &= 255;
    num = fread(&dimensionality, 1, 1, stdin);
    assert(1 == num);
    dimensionality &= 255;
    num = fread(&doubles, 4, 1, stdin);
    assert(1 == num);

    Decompress(blocks, warpsperblock, dimensionality, doubles);
  }
  else {
    fprintf(stderr, "usage:\n");
    fprintf(stderr, "compress: %s blocks warps/block (dimensionality) < file.in > file.gfc\n", argv[0]);
    fprintf(stderr, "decompress: %s < file.gfc > file.out\n", argv[0]);
  }

  return 0;
}
