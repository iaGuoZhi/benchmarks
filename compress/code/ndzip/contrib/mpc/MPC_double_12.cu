#include "hip/hip_runtime.h"
/*
MPC code [double] (LnVs BIT LVs ZE): A GPU-based compressor for arrays of 
double-precision floating-point values.  See the following publication for
more information: http://cs.txstate.edu/~mb92/papers/cluster15.pdf.

Copyright (c) 2015-2020, Texas State University. All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.
   * Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the following disclaimer in the
     documentation and/or other materials provided with the distribution.
   * Neither the name of Texas State University nor the names of its
     contributors may be used to endorse or promote products derived from
     this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL TEXAS STATE UNIVERSITY BE LIABLE FOR ANY
DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Authors: Annie Yang and Martin Burtscher

URL: The latest version of this code is available at
https://userweb.cs.txstate.edu/~burtscher/research/MPC/.

Publication: This work is described in detail in the following paper.
Annie Yang, Hari Mukka, Farbod Hesaaraki, and Martin Burtscher. MPC: A
Massively Parallel Compression Algorithm for Scientific Data. Proceedings
of the IEEE International Conference on Cluster Computing, pp. 381-389.
September 2015.
*/


#include "MPC_12.h"
#include <cstdio>
#include <cassert>

#ifdef __cplusplus
extern "C" {
#endif

#define TPB 1024  /* do not change */

// BLOCKS_PER_SM on the device must be equal to blocksPerSM (see below) on the host
#ifdef __CUDA_ARCH__
#   if __CUDA_ARCH__ == 750 || __CUDA_ARCH__ == 860
#      define BLOCKS_PER_SM 1
#   elif __CUDA_ARCH__ <= 860
#      define BLOCKS_PER_SM 2
#   else
#      error Check how many 1024-thread blocks per SM your target supports and update BLOCKS_PER_SM
#   endif
#else
#   define BLOCKS_PER_SM 0 // dummy for host compilation
#endif

#if __CUDA_ARCH__ >= 700 || CUDART_VERSION >= 9000
// Unsynced versions are deprecated
#define __shfl(...) __shfl_sync(0xffffffff, __VA_ARGS__)
#define __shfl_up(...) __shfl_up_sync(0xffffffff, __VA_ARGS__)
#define __ballot(...) __ballot_sync(0xffffffff, __VA_ARGS__)
#endif

static inline __device__
void prefixsum(int &val, int sbuf[TPB])
{
  const int warp = threadIdx.x >> 5;
  const int lane = threadIdx.x & 31;

  for (int d = 1; d < 32; d *= 2) {
    int tmp = __shfl_up(val, d);
    if (lane >= d) val += tmp;
  }
  if (lane == 31) sbuf[warp] = val;

  __syncthreads();
  if (warp == 0) {
    int v = sbuf[lane];
    for (int d = 1; d < 32; d *= 2) {
      int tmp = __shfl_up(v, d);
      if (lane >= d) v += tmp;
    }
    sbuf[lane] = v;
  }

  __syncthreads();
  if (warp > 0) {
    val += sbuf[warp - 1];
  }
}

static inline __device__
void prefixsumlong(long &val, long sbuf[TPB])
{
  const int warp = threadIdx.x >> 5;
  const int lane = threadIdx.x & 31;

  for (int d = 1; d < 32; d *= 2) {
    unsigned int tmpl = __shfl_up((int)val, d);
    long tmph = __shfl_up((int)(val >> 32), d);
    if (lane >= d) val += (tmph << 32) + tmpl;
  }
  if (lane == 31) sbuf[warp] = val;

  __syncthreads();
  if (warp == 0) {
    long v = sbuf[lane];
    for (int d = 1; d < 32; d *= 2) {
      unsigned int tmpl = __shfl_up((int)v, d);
      long tmph = __shfl_up((int)(v >> 32), d);
      if (lane >= d) v += (tmph << 32) + tmpl;
    }
    sbuf[lane] = v;
  }

  __syncthreads();
  if (warp > 0) {
    val += sbuf[warp - 1];
  }
}

static inline __device__
void prefixsumdimlong(long &val, long sbuf[TPB], const unsigned char dim)
{
  const int tid = threadIdx.x;
  const int warp = tid >> 5;
  const int lane = tid & 31;
  const int tix = (warp * dim) + (tid % dim);

  for (int d = dim; d < 32; d *= 2) {
    unsigned int tmpl = __shfl_up((int)val, d);
    long tmph = __shfl_up((int)(val >> 32), d);
    if (lane >= d) val += (tmph << 32) + tmpl;
  }
  if ((lane + dim) > 31) sbuf[tix] = val;

  __syncthreads();
  if (warp < dim) {
    const int idx = (lane * dim) + warp;
    long v = sbuf[idx];
    for (int d = 1; d < 32; d *= 2) {
      unsigned int tmpl = __shfl_up((int)v, d);
      long tmph = __shfl_up((int)(v >> 32), d);
      if (lane >= d) v += (tmph << 32) + tmpl;
    }
    sbuf[idx] = v;
  }

  __syncthreads();
  if (warp > 0) {
    val += sbuf[tix - dim];
  }
}

/*****************************************************************************
This is the GPU compression kernel, which requires 1024 threads per block and
should be launched with as many blocks as the GPU can run simultaneously.

Inputs
------
n: the number of double values to be compressed
original: the input array holding the n doubles (has to be cast to a long array)
goffset: a temporary array with m elements where m = number of thread blocks
dim: the dimensionality of the input data (dim must be between 1 and 32)

Output
------
compressed: the output array that holds the compressed data in integer format

The output array needs to provide space for up to 2 + n + (n + 31) / 32 elements.
The upper half of the first element specifies how many elements are actually
used.  It should be replaced by the value n before the data is further processed.
*****************************************************************************/

static __global__ __launch_bounds__(TPB, BLOCKS_PER_SM)
void MPCcompress(const int n, long* const original, long* const compressed, volatile int* const goffset, unsigned char dim)
{
  const int tid = threadIdx.x;
  const int tidm1 = tid - 1;
  const int tidmdim = tid - dim;
  const int lanex = tid & 63;
  const int warpx = tid & 0x3c0;
  const int bid = blockIdx.x;
  const int gdim = gridDim.x;
  const int bid1 = ((bid + 1) == gdim) ? 0 : (bid + 1);
  const int init = 1 + (n + 63) / 64;
  const int chunksm1 = ((n + (TPB - 1)) / TPB) - 1;

  __shared__ int start, top;
  __shared__ long sbuf1[TPB], sbuf2[TPB];

  for (int chunk = bid; chunk <= chunksm1; chunk += gdim) {
    const int idx = tid + chunk * TPB;

    long v1 = 0;
    if (idx < n) {
      v1 = original[idx];
      sbuf1[tid] = v1;
    }

    __syncthreads();
    if (tid >= dim) {
      if (idx < n) {
        v1 -= sbuf1[tidmdim];
      }
    }
    sbuf2[tid] = v1;

    __syncthreads();
    long v2 = 0;

    for (int i = 63; i >= 0; i--) {
      v2 = (v2 << 1) + ((sbuf2[warpx + i] >> lanex) & 1);
    }
    sbuf1[tid] = v2;

    __syncthreads();
    if (tid > 0) {
      v2 -= sbuf1[tidm1];
    }

    int loc = 0;
    if (v2 != 0) loc = 1;

    unsigned int bitmap = __ballot(loc);

    if (lanex == 32) {
      sbuf2[tid] = bitmap;
    }

    __syncthreads();
    if (lanex == 0) {
      if (idx < n) compressed[1 + idx / 64] = (sbuf2[tid + 32] << 32) + bitmap;
    }

    prefixsum(loc, (int*)sbuf1);

    if (v2 != 0) {
      sbuf2[loc - 1] = v2;
    }

    if (tid == (TPB - 1)) {
      int st = init;
      if (chunk > 0) {
        do {
          st = goffset[bid];
        } while (st < 0);  // busy waiting
      }
      goffset[bid1] = st + loc;
      goffset[bid] = -1;
      if (chunk == chunksm1) {
        compressed[0] = (((long)(st + loc)) << 32) + (0x43504d00 - 1) + dim;
      }
      top = loc;
      start = st;
    }

    __syncthreads();
    if (tid < top) {
      compressed[start + tid] = sbuf2[tid];
    }
  }
}

/*****************************************************************************
This is the GPU decompression kernel, which requires 1024 threads per block
and should be launched with as many blocks as the GPU can run simultaneously.

Inputs
------
compressed: the input array holding the compressed data in integer format
goffset: a temporary array with m elements where m = number of thread blocks

The upper half of the first element must hold the value n, i.e., the number
of doubles that the data will generate upon decompression.

Output
------
decompressed: the output array holding the decompressed data in integer format

The output array needs to provide space for n elements has to be cast to an
array of doubles before it can be used.
*****************************************************************************/

static __global__ __launch_bounds__(TPB, BLOCKS_PER_SM)
void MPCdecompress(long* const compressed, long* const decompressed, volatile int* const goffset)
{
  const int dim = (compressed[0] & 31) + 1;
  const int n = compressed[0] >> 32;
  const int tid = threadIdx.x;
  const int lanex = tid & 63;
  const int warpx = tid & 0x3c0;
  const int bid = blockIdx.x;
  const int gdim = gridDim.x;
  const int bid1 = ((bid + 1) == gdim) ? 0 : (bid + 1);
  const int init = 1 + (n + 63) / 64;
  const int nru = (n - 1) | 63;
  const int chunksm1 = ((n + (TPB - 1)) / TPB) - 1;

  __shared__ int start, top;
  __shared__ long sbuf1[TPB], sbuf2[TPB];

  for (int chunk = bid; chunk <= chunksm1; chunk += gdim) {
    const int idx = tid + chunk * TPB;

    int flag = 0;
    if (idx <= nru) {
      flag = (compressed[1 + idx / 64] >> lanex) & 1;
    }
    int loc = flag;

    __syncthreads();
    prefixsum(loc, (int*)sbuf1);

    if (tid == (TPB - 1)) {
      int st = init;
      if (chunk > 0) {
        do {
          st = goffset[bid];
        } while (st < 0);  // busy waiting
      }
      goffset[bid1] = st + loc;
      goffset[bid] = -1;
      top = loc;
      start = st;
    }

    __syncthreads();
    if (tid < top) {
      sbuf2[tid] = compressed[start + tid];
    }

    __syncthreads();
    long v2 = 0;
    if (flag != 0) {
      v2 = sbuf2[loc - 1];
    }

    prefixsumlong(v2, sbuf1);

    sbuf2[tid] = v2;

    __syncthreads();
    long v1 = 0;
    for (int i = 63; i >= 0; i--) {
      v1 = (v1 << 1) + ((sbuf2[warpx + i] >> lanex) & 1);
    }

    prefixsumdimlong(v1, sbuf1, dim);

    if (idx < n) {
      decompressed[idx] = v1;
    }
  }
}

static void CudaTest(const char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    abort();
  }
}

int MPC_double_compressBound(int insize)
{
    return insize + 1 + (insize + 63) / 64;
}

static_assert(sizeof(long) == sizeof(double));

int MPC_double_compressMemory(long *output, const long *input, int insize, int dim,
        uint64_t *kernel_time_us)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
        fprintf(stderr, "There is no CUDA capable device\n");
        abort();
    }
    if (deviceProp.major < 3) {
        fprintf(stderr, "Need at least compute capability 3.0\n");
        abort();
    }
    const int blocksPerSM = deviceProp.maxThreadsPerMultiProcessor / TPB;
    const int blocks = deviceProp.multiProcessorCount * blocksPerSM;

    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    int outsize = MPC_double_compressBound(insize);
    assert(0 < dim);  assert(dim <= 32);

    long* d_in, * d_out;
    int* d_offs;
    hipMalloc(&d_in, insize * sizeof(long));  CudaTest("malloc failed");
    hipMalloc(&d_out, outsize * sizeof(long));  CudaTest("malloc failed");
    hipMalloc(&d_offs, blocks * sizeof(int));  CudaTest("malloc failed");

    hipMemcpy(d_in, input, insize * sizeof(long), hipMemcpyHostToDevice);  CudaTest("memcpy failed");

    hipEvent_t begin, end;
    if (kernel_time_us) {
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, NULL);
    }

    hipMemset(d_offs, -1, blocks * sizeof(int));
    MPCcompress<<<blocks, TPB>>>(insize, d_in, d_out, d_offs, dim);

    if (kernel_time_us) {
        hipEventRecord(end, NULL);
        float duration_ms;
        hipEventSynchronize(end);
        hipEventElapsedTime(&duration_ms, begin, end);
        *kernel_time_us = (uint64_t) (duration_ms * 1000);
        hipEventDestroy(end);
        hipEventDestroy(begin);
    }

    hipDeviceSynchronize();
    CudaTest("compression failed");

    hipMemcpy(output, d_out, sizeof(long), hipMemcpyDeviceToHost);  CudaTest("memcpy failed");
    outsize = output[0] >> 32;

    hipMemcpy(output, d_out, outsize * sizeof(long), hipMemcpyDeviceToHost);  CudaTest("memcpy failed");
    output[0] = (((long)insize) << 32) + (0x43504d00 - 1) + dim;

    hipFree(d_offs);
    hipFree(d_out);
    hipFree(d_in);
    CudaTest("free failed");

    return outsize;
}

int MPC_double_decompressedSize(const long *input, int insize) {
    assert(insize > 0);
    assert(((input[0] >> 8) & 0xffffff) == 0x43504d);
    return input[0] >> 32;
}

int MPC_double_decompressMemory(long *output, const long *input, int insize,
        uint64_t *kernel_time_us)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
        fprintf(stderr, "There is no CUDA capable device\n");
        abort();
    }
    if (deviceProp.major < 3) {
        fprintf(stderr, "Need at least compute capability 3.0\n");
        abort();
    }
    const int blocksPerSM = deviceProp.maxThreadsPerMultiProcessor / TPB;
    const int blocks = deviceProp.multiProcessorCount * blocksPerSM;

    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

    int outsize = MPC_double_decompressedSize(input, insize);

    long* d_in, * d_out;
    int* d_offs;
    hipMalloc(&d_in, insize * sizeof(long));  CudaTest("malloc failed");
    hipMalloc(&d_out, outsize * sizeof(long));  CudaTest("malloc failed");
    hipMalloc(&d_offs, blocks * sizeof(int));  CudaTest("malloc failed");

    hipMemcpy(d_in, input, insize * sizeof(long), hipMemcpyHostToDevice);  CudaTest("memcpy failed");

    hipEvent_t begin, end;
    if (kernel_time_us) {
        hipEventCreate(&begin);
        hipEventCreate(&end);
        hipEventRecord(begin, NULL);
    }

    hipMemset(d_offs, -1, blocks * sizeof(int));
    MPCdecompress<<<blocks, TPB>>>(d_in, d_out, d_offs);
    CudaTest("decompression failed");

    if (kernel_time_us) {
        hipEventRecord(end, NULL);
        float duration_ms;
        hipEventSynchronize(end);
        hipEventElapsedTime(&duration_ms, begin, end);
        *kernel_time_us = (uint64_t) (duration_ms * 1000);
        hipEventDestroy(end);
        hipEventDestroy(begin);
    }

    hipDeviceSynchronize();
    hipMemcpy(output, d_out, outsize * sizeof(long), hipMemcpyDeviceToHost);  CudaTest("memcpy failed");

    hipFree(d_offs);
    hipFree(d_out);
    hipFree(d_in);
    CudaTest("free failed");

    return outsize;
}

#ifdef __cplusplus
}
#endif
