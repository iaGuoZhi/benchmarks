#include "hip/hip_runtime.h"
#include "test_utils.hh"

#include <ndzip/cuda_bits.cuh>

using namespace ndzip;
using namespace ndzip::detail;
using namespace ndzip::detail::gpu_cuda;


template<typename T>
struct logical_or {
    __host__ __device__ T operator()(T a, T b) const { return a || b; }
};


__global__ void test_warp_reduce(index_type *out) {
    out[threadIdx.x] = warp_reduce(index_type{1}, plus<index_type>{});
}

__global__ void test_warp_inclusive_scan(index_type *out) {
    out[threadIdx.x] = warp_inclusive_scan(static_cast<index_type>(threadIdx.x), plus<index_type>{});
}

__global__ void test_warp_exclusive_scan(index_type *out) {
    out[threadIdx.x] = warp_exclusive_scan(static_cast<index_type>(threadIdx.x), index_type{0}, plus<index_type>{});
}

template<typename T, index_type Range, index_type GroupSize>
__global__ void test_inclusive_scan(T *out) {
    auto block = known_size_block<GroupSize>{};
    distribute_for(Range, block, [&](index_type item) { out[item] = 1; });
    __syncthreads();
    inclusive_scan<Range>(block, out, plus<T>{});
}


TEST_CASE("Warp-cooperative primitives work", "[cuda][scan]") {
    cuda_buffer<index_type> buf(2 * warp_size);
    std::vector<index_type> out(2 * warp_size);
    std::vector<index_type> iota(2 * warp_size);
    std::vector<index_type> ref(2 * warp_size);

    std::iota(iota.begin(), iota.end(), index_type{0});

    SECTION("warp_reduce") {
        test_warp_reduce<<<1, 2 * warp_size>>>(buf.get());
        CHECKED_CUDA_CALL(
                hipMemcpy, out.data(), buf.get(), 2 * warp_size * sizeof(index_type), hipMemcpyDeviceToHost);
        std::fill(ref.begin(), ref.end(), warp_size);
        CHECK_FOR_VECTOR_EQUALITY(out, ref);
    }

    SECTION("warp_inclusive_scan") {
        test_warp_inclusive_scan<<<1, 2 * warp_size>>>(buf.get());
        CHECKED_CUDA_CALL(
                hipMemcpy, out.data(), buf.get(), 2 * warp_size * sizeof(index_type), hipMemcpyDeviceToHost);
        iter_inclusive_scan(iota.begin(), iota.begin() + warp_size, ref.begin(), plus<index_type>{});
        iter_inclusive_scan(
                iota.begin() + warp_size, iota.begin() + 2 * warp_size, ref.begin() + warp_size, plus<index_type>{});
        CHECK_FOR_VECTOR_EQUALITY(out, ref);
    }

    SECTION("warp_exclusive_scan") {
        test_warp_exclusive_scan<<<1, 2 * warp_size>>>(buf.get());
        CHECKED_CUDA_CALL(
                hipMemcpy, out.data(), buf.get(), 2 * warp_size * sizeof(index_type), hipMemcpyDeviceToHost);
        iter_exclusive_scan(iota.begin(), iota.begin() + warp_size, ref.begin(), plus<index_type>{});
        iter_exclusive_scan(
                iota.begin() + warp_size, iota.begin() + 2 * warp_size, ref.begin() + warp_size, plus<index_type>{});
        CHECK_FOR_VECTOR_EQUALITY(out, ref);
    }
}


TEMPLATE_TEST_CASE("Subgroup hierarchical inclusive scan works", "[cuda][scan]", uint32_t, uint64_t) {
    constexpr index_type group_size = 512;
    constexpr index_type n_groups = 9;
    constexpr index_type range = group_size * n_groups;

    cuda_buffer<TestType> out(range);
    test_inclusive_scan<TestType, range, group_size><<<1, group_size>>>(out.get());

    std::vector<TestType> cpu_input(range, TestType{1});
    std::vector<TestType> cpu_result(range);
    iter_inclusive_scan(cpu_input.begin(), cpu_input.end(), cpu_result.begin());

    std::vector<TestType> gpu_result(range);
    CHECKED_CUDA_CALL(hipMemcpy, gpu_result.data(), out.get(), out.size() * sizeof(TestType), hipMemcpyDeviceToHost);

    CHECK_FOR_VECTOR_EQUALITY(cpu_result, gpu_result);
}


TEMPLATE_TEST_CASE("hierarchical_inclusive_scan produces the expected results", "[cuda][scan]", plus<uint32_t>,
        logical_or<uint32_t>) {
    std::vector<uint32_t> input(size_t{1} << 24u);
    std::iota(input.begin(), input.end(), uint32_t{});

    std::vector<uint32_t> cpu_prefix_sum(input.size());
    iter_inclusive_scan(input.begin(), input.end(), cpu_prefix_sum.begin(), TestType{});

    cuda_buffer<uint32_t> prefix_sum_buf(input.size());
    CHECKED_CUDA_CALL(hipMemcpy, prefix_sum_buf.get(), input.data(), prefix_sum_buf.size() * sizeof(uint32_t),
            hipMemcpyHostToDevice);

    auto intermediate_bufs = hierarchical_inclusive_scan_allocate<uint32_t>(prefix_sum_buf.size());
    hierarchical_inclusive_scan(prefix_sum_buf.get(), intermediate_bufs, prefix_sum_buf.size(), TestType{});

    std::vector<uint32_t> gpu_prefix_sum(input.size());
    CHECKED_CUDA_CALL(hipMemcpy, gpu_prefix_sum.data(), prefix_sum_buf.get(), prefix_sum_buf.size() * sizeof(uint32_t),
            hipMemcpyDeviceToHost);

    CHECK_FOR_VECTOR_EQUALITY(gpu_prefix_sum, cpu_prefix_sum);
}
